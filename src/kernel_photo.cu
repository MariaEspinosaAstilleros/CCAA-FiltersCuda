#include "hip/hip_runtime.h"
#include <opencv4/opencv2/imgcodecs.hpp>
#include <opencv4/opencv2/core/utility.hpp>
#include <opencv4/opencv2/highgui.hpp>
#include <opencv4/opencv2/imgproc/imgproc.hpp>

#include <../include/kernel_photo.h>
#include <../include/colors.h>

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

#define BLOCK_SIZE 32
#define GRID_SIZE 128

/*Kernel*/
__global__ void kernel_convolution_sobel(unsigned char* src_img, unsigned char* dst_img, int width_img, int height_img){

    //Gradients of the sobel filter
    int gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int gy[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
    
    int num_row = blockIdx.x * blockDim.x + threadIdx.x;
    int num_col = blockIdx.y * blockDim.y + threadIdx.y;

    int index = num_row * width_img + num_col;

    if(num_col < (width_img - 1) && num_row < (height_img - 1)){
        float grad_x= (src_img[index] * gx[0][0]) + (src_img[index+1] * gx[0][1]) + (src_img[index+2] * gx[0][2]) +
                      (src_img[index] * gx[1][0]) + (src_img[index+1] * gx[1][1]) + (src_img[index+2] * gx[1][2]) +
                      (src_img[index] * gx[2][0]) + (src_img[index+1] * gx[2][1]) + (src_img[index+2] * gx[2][2]);

        float grad_y= (src_img[index] * gy[0][0]) + (src_img[index+1] * gy[0][1]) + (src_img[index+2] * gy[0][2]) +
                      (src_img[index] * gy[1][0]) + (src_img[index+1] * gy[1][1]) + (src_img[index+2] * gy[1][2]) +
                      (src_img[index] * gy[2][0]) + (src_img[index+1] * gy[2][1]) + (src_img[index+2] * gy[2][2]);

        float gradient = sqrtf(grad_x * grad_x + grad_y * grad_y);

        if(gradient > 255)gradient = 255;
        if(gradient < 0)gradient = 0;

        dst_img[index] = gradient;
    }
}

/*Main*/
int main(int argc, char **argv){ 
    int option;
    std::cout << "Select an option: photo " << MAGENTA << "(1) " << RESET", camera "  << MAGENTA << "(2) " << RESET "or video " << MAGENTA << "(3) " << RESET << std::endl;
    std::cin >> option;

    switch(option){
        case 1: 
            optionPhoto();
            break;
        case 2: 
            optionCamera();
            break;
        case 3: //video
            optionVideo();
            break;
    }
    return 0;
}

/* Auxiliar functions*/
hipError_t testCuErr(hipError_t dst_img){
    if (dst_img != hipSuccess) {
        printf("CUDA Runtime Error: %s\n", 
            hipGetErrorString(dst_img));
        assert(dst_img == hipSuccess);
    }
    return dst_img;
}

void optionPhoto(){
    std::string input_img_path;
    std::cout << "Select a photo to apply the sobel filter:" << std::endl;
    std::cin >> input_img_path;
    cv::Mat src_img = cv::imread(input_img_path, cv::IMREAD_GRAYSCALE);
   
    if(!src_img.data){
        std::cerr << "ERROR. No image data." << std::endl;
        std::cout << "Enter path that contains the image: " << YELLOW << "img/<name_image>" << RESET << std::endl;
        exit(-1);
    }else{
        sobelFilter(&src_img); // apply sobel filter to the photo

        cv::resize(src_img, src_img, cv::Size(1366,768));
        cv::imshow("CUDA Sobel", src_img);
        cv::waitKey(0);
    }
}

void optionCamera(){
    cv::VideoCapture camera(0);

    if (!camera.isOpened()) {
        std::cerr << "ERROR: Could not open camera" << std::endl;
        exit(-1);
    }

    while (true){ 
        cv::Mat cam_frame;
        camera.read(cam_frame);
        cv::imshow("CUDA Sobel WebCam", cam_frame);
        if (cv::waitKey(10) >= 0)
        break;
    }
}

void optionVideo(){

}

void sobelFilter(cv::Mat *src_img){

    hipFree(0);
    unsigned char *dev_src, *dev_sobel;
    int img_size = src_img->rows * src_img->cols * sizeof(unsigned char);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(GRID_SIZE, GRID_SIZE);

    testCuErr(hipMalloc((void**)&dev_src, img_size));
    testCuErr(hipMalloc((void**)&dev_sobel, img_size));

    //copy data to GPU
    testCuErr(hipMemcpy(dev_src, src_img->data, img_size, hipMemcpyHostToDevice));

    //start time
    auto start = std::chrono::high_resolution_clock::now();

    //call kernel
    kernel_convolution_sobel<<<numBlocks,threadsPerBlock>>>(dev_src, dev_sobel, src_img->cols, src_img->rows);
    testCuErr(hipGetLastError()); testCuErr(hipDeviceSynchronize());

    //end time 
    auto end = std::chrono::high_resolution_clock::now(); 

    //diff time
    std::chrono::duration<double> diff = end - start;
    std::cout << "Elapsed time: " << diff.count() << " seg" << std::endl;

    //copy data to CPU
    testCuErr(hipMemcpy(src_img->data, dev_sobel, img_size, hipMemcpyDeviceToHost));

    //free mem in device
    testCuErr(hipFree(dev_src)); testCuErr(hipFree(dev_sobel));
}
