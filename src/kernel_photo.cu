#include "hip/hip_runtime.h"
#include <opencv4/opencv2/imgcodecs.hpp>
#include <opencv4/opencv2/core/utility.hpp>
#include <opencv4/opencv2/highgui.hpp>
#include <opencv4/opencv2/imgproc/imgproc.hpp>

#include <../include/kernel_photo.h>
#include <../include/Filter.h>
#include <../include/colors.h>

#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

#define BLOCK_SIZE 32
#define GRID_SIZE 128

/*Kernel*/
__global__ void kernelConvolutionSobel(unsigned char* src_img, unsigned char* dst_img, int width_img, int height_img){

    //Gradients of the sobel filter
    int gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int gy[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
    
    int num_row = blockIdx.x * blockDim.x + threadIdx.x;
    int num_col = blockIdx.y * blockDim.y + threadIdx.y;

    int index = num_row * width_img + num_col;

    if(num_col < (width_img - 1) && num_row < (height_img - 1)){
        float grad_x= (src_img[index] * gx[0][0]) + (src_img[index+1] * gx[0][1]) + (src_img[index+2] * gx[0][2]) +
                      (src_img[index] * gx[1][0]) + (src_img[index+1] * gx[1][1]) + (src_img[index+2] * gx[1][2]) +
                      (src_img[index] * gx[2][0]) + (src_img[index+1] * gx[2][1]) + (src_img[index+2] * gx[2][2]);

        float grad_y= (src_img[index] * gy[0][0]) + (src_img[index+1] * gy[0][1]) + (src_img[index+2] * gy[0][2]) +
                      (src_img[index] * gy[1][0]) + (src_img[index+1] * gy[1][1]) + (src_img[index+2] * gy[1][2]) +
                      (src_img[index] * gy[2][0]) + (src_img[index+1] * gy[2][1]) + (src_img[index+2] * gy[2][2]);

        float gradient = sqrtf(grad_x * grad_x + grad_y * grad_y);

        if(gradient > 255)gradient = 255;
        if(gradient < 0)gradient = 0;

        dst_img[index] = gradient;
    }
}

__host__ void Filter::sobel(cv::Mat *src_img){
    hipFree(0);
    unsigned char *dev_src, *dev_sobel;
    int img_size = src_img->rows * src_img->cols * sizeof(unsigned char);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(GRID_SIZE, GRID_SIZE);

    testCuErr(hipMalloc((void**)&dev_src, img_size));
    testCuErr(hipMalloc((void**)&dev_sobel, img_size));

    //copy data to GPU
    testCuErr(hipMemcpy(dev_src, src_img->data, img_size, hipMemcpyHostToDevice));

    //start time
    auto start = std::chrono::high_resolution_clock::now();

    //call kernel
    kernelConvolutionSobel<<<numBlocks,threadsPerBlock>>>(dev_src, dev_sobel, src_img->cols, src_img->rows);
    testCuErr(hipGetLastError()); testCuErr(hipDeviceSynchronize());

    //end time 
    auto end = std::chrono::high_resolution_clock::now(); 

    //diff time
    std::chrono::duration<double> diff = end - start;
    std::cout << "Elapsed time: " << diff.count() << " seg" << std::endl;

    //copy data to CPU
    testCuErr(hipMemcpy(src_img->data, dev_sobel, img_size, hipMemcpyDeviceToHost));

    //free mem in device
    testCuErr(hipFree(dev_src)); testCuErr(hipFree(dev_sobel));
}

/*__host__ void Filter::other(){

}*/